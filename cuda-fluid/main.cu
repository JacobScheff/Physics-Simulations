
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <vector>

#define SCREEN_SIZE_X 1200
#define SCREEN_SIZE_Y 600
#define GRID_SIZE_X 80
#define GRID_SIZE_Y 40
int const SCREEN_SIZE_C[2] = {SCREEN_SIZE_X, SCREEN_SIZE_Y}; // The size of the screen
int const GRID_SIZE_C[2] = {GRID_SIZE_X, GRID_SIZE_Y};     // How many grid cells to divide the screen into

int const TIME_BETWEEN_FRAMES = 2;
float const PARTICLE_RADIUS = 1.25;                                // The radius of the particles
int const PARTICLE_AMOUNT_X = 192;                                 // The number of particles in the x direction
int const PARTICLE_AMOUNT_Y = 96;                                  // The number of particles in the y direction
int const PARTICLE_AMOUNT = PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; // The total number of particles
float const PADDING = 50.0;                                        // The padding around the screen

#define RADIUS_OF_INFLUENCE (75.0 / 4.0) // The radius of the sphere of influence. Also the radius to search for particles to calculate the density
#define TARGET_DENSITY 0.2; // The target density of the fluid
#define PRESSURE_MULTIPLIER 500.0; // The multiplier for the pressure force
#define GRAVITY 0.2; // The strength of gravity
#define LOOK_AHEAD_TIME (1.0 / 60.0); // The time to look ahead when calculating the predicted position
#define VISCOSITY 0.1; // The viscosity of the fluid
#define DAMPENING 0.95; // How much to slow down particles when they collide with the walls
#define dt (1.0 / 8.0); // The time step

int const GRIDS_TO_CHECK[2] = {int(RADIUS_OF_INFLUENCE / SCREEN_SIZE_C[0] * GRID_SIZE_C[0] + 1.0), int(RADIUS_OF_INFLUENCE / SCREEN_SIZE_C[1] * GRID_SIZE_C[1] + 1.0)}; // How many grid cells to check in each direction

// Grid functions
__device__ __host__
int* pos_to_grid(float x, float y)
{
  int grid[2] =  {
      (int)fmax(fmin(floor(x / SCREEN_SIZE_X * GRID_SIZE_X), GRID_SIZE_X - 1), 0),
      (int)fmax(fmin(floor(y / SCREEN_SIZE_Y * GRID_SIZE_Y), GRID_SIZE_Y - 1), 0)};
  return grid;
}

__device__ __host__
int grid_to_index(int x, int y)
{
  return y * GRID_SIZE_X + x;
}

// Kernel function to calculate densities
__global__ void calculate_densities(float **positions, float *densities, float *radii, int *particle_lookup, int *particle_counts, int GRIDS_TO_CHECK_X, int GRIDS_TO_CHECK_Y, int particle_amount)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= PARTICLE_AMOUNT)
    return;

  int* grid = pos_to_grid(positions[index][0], positions[index][1]);
  float density = 0.0;

  for(int g = 0; g < (GRIDS_TO_CHECK_X * 2 + 1) * (GRIDS_TO_CHECK_Y * 2 + 1); g++){
    int gx = g / (GRIDS_TO_CHECK_Y * 2 + 1) - GRIDS_TO_CHECK_X;
    int gy = g % (GRIDS_TO_CHECK_Y * 2 + 1) - GRIDS_TO_CHECK_Y;

    if(grid[0] + gx < 0 || grid[0] + gx >= GRID_SIZE_X || grid[1] + gy < 0 || grid[1] + gy >= GRID_SIZE_Y){
      continue;
    }

    int first_grid_index = grid_to_index(grid[0] + gx, grid[1] + gy);
    if(first_grid_index < 0 || first_grid_index >= GRID_SIZE_X * GRID_SIZE_Y){
      continue;
    }

    int starting_index = particle_lookup[first_grid_index];
    if(starting_index == -1){
      continue;
    }

    int ending_index = starting_index + particle_counts[first_grid_index];

    for(int i = starting_index; i <= ending_index; i++){
      
    }
  }

  densities[index] = density;
}

void sort(float **positions, float **velocities, float *radii, float *densities, float **pressure_force, float **viscosity_force, int *particle_lookup, int *particle_counts)
{
  // Map all particles to their grid cell
  std::vector<std::vector<std::vector<int>>> index_map(GRID_SIZE_C[0], std::vector<std::vector<int>>(GRID_SIZE_C[1], std::vector<int>()));
  for (int i = 0; i < PARTICLE_AMOUNT; i++)
  {
    int* grid = pos_to_grid(positions[i][0], positions[i][1]);
    index_map[grid[0]][grid[1]].push_back(i);
  }

  // Create a new list of particles
  float **new_positions = new float *[PARTICLE_AMOUNT];
  float **new_velocities = new float *[PARTICLE_AMOUNT];
  float *new_radii = new float[PARTICLE_AMOUNT];
  float *new_densities = new float[PARTICLE_AMOUNT];
  float **new_pressure_force = new float *[PARTICLE_AMOUNT];
  float **new_viscosity_force = new float *[PARTICLE_AMOUNT];

  // Iterate over all grid cells
  for (int i = 0; i < GRID_SIZE_C[0]; i++)
  {
    for (int j = 0; j < GRID_SIZE_C[1]; j++)
    {
      int grid_index = i + j * GRID_SIZE_C[0];
      int index = -1;

      // Iterate over all particles in the grid cell
      for (int k = 0; k < index_map[i][j].size(); k++)
      {
        int particle_index = index_map[i][j][k];
        new_positions[particle_index] = positions[particle_index];
        new_velocities[particle_index] = velocities[particle_index];
        new_radii[particle_index] = radii[particle_index];
        new_densities[particle_index] = densities[particle_index];
        new_pressure_force[particle_index] = pressure_force[particle_index];
        new_viscosity_force[particle_index] = viscosity_force[particle_index];

        if (index == -1)
        {
          index = particle_index;
        }
        particle_counts[grid_index]++;
      }

      particle_lookup[grid_index] = index;
    }
  }

  positions = new_positions;
  velocities = new_velocities;
  radii = new_radii;
  densities = new_densities;
  pressure_force = new_pressure_force;
  viscosity_force = new_viscosity_force;

  delete[] new_positions;
  delete[] new_velocities;
  delete[] new_radii;
  delete[] new_densities;
  delete[] new_pressure_force;
  delete[] new_viscosity_force;
}

int main(void)
{
  // Initialize data
  float **positions, **velocities, **pressure_force, **viscosity_force;
  float *densities, *radii;
  int *particle_lookup, *particle_counts;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&positions, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&velocities, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&pressure_force, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&viscosity_force, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&densities, PARTICLE_AMOUNT * sizeof(float));
  hipMallocManaged(&radii, PARTICLE_AMOUNT * sizeof(float));
  hipMallocManaged(&particle_lookup, GRID_SIZE_C[0] * GRID_SIZE_C[1] * sizeof(int));
  hipMallocManaged(&particle_counts, GRID_SIZE_C[0] * GRID_SIZE_C[1] * sizeof(int));

  // Initialize data
  for (int i = 0; i < PARTICLE_AMOUNT; i++)
  {
    positions[i] = new float[2];
    velocities[i] = new float[2];
    pressure_force[i] = new float[2];
    viscosity_force[i] = new float[2];
    positions[i][0] = (i + 0.5) * (SCREEN_SIZE_C[0] - 2.0 * PADDING) / PARTICLE_AMOUNT_X + PADDING;
    positions[i][1] = (i + 0.5) * (SCREEN_SIZE_C[1] - 2.0 * PADDING) / PARTICLE_AMOUNT_Y + PADDING;
    velocities[i][0] = 0.0;
    velocities[i][1] = 0.0;
    densities[i] = 0.0;
    radii[i] = PARTICLE_RADIUS;
    pressure_force[i][0] = 0.0;
    pressure_force[i][1] = 0.0;
    viscosity_force[i][0] = 0.0;
    viscosity_force[i][1] = 0.0;

    if (i < GRID_SIZE_C[0] * GRID_SIZE_C[1])
    {
      particle_lookup[i] = -1;
      particle_counts[i] = 0;
    }
  }
  
  // Sort the particles
  sort(positions, velocities, radii, densities, pressure_force, viscosity_force, particle_lookup, particle_counts);

  // Get start time
  auto start = std::chrono::high_resolution_clock::now();

  // Get the number of blocks and threads
  int blockSize = 256;
  int numBlocks = (PARTICLE_AMOUNT + blockSize - 1) / blockSize;

  // Calculate densities
  calculate_densities<<<numBlocks, blockSize>>>(positions, densities, radii, particle_lookup, particle_counts, GRIDS_TO_CHECK[0], GRIDS_TO_CHECK[1], PARTICLE_AMOUNT);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Print end time in ms
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> elapsed = end - start;
  std::cout << "Elapsed time in milliseconds : " << elapsed.count() << " ms" << std::endl;

  // Free memory
  hipFree(positions);
  hipFree(velocities);
  hipFree(pressure_force);
  hipFree(viscosity_force);
  hipFree(densities);
  hipFree(radii);
  hipFree(particle_lookup);
  hipFree(particle_counts);

  std::cout << "Hello, World!" << std::endl;

  return 0;
}

__device__
float density_to_pressure(float density)
{
  float density_error = density - TARGET_DENSITY;
  return density_error * PRESSURE_MULTIPLIER;
}

__device__
float smoothing_kernel(float distance)
{
  if (distance >= RADIUS_OF_INFLUENCE)
  {
    return 0.0;
  }

  float volume = 3.141592653589 * pow(RADIUS_OF_INFLUENCE, 4.0) / 6.0;
  return (RADIUS_OF_INFLUENCE - distance) * (RADIUS_OF_INFLUENCE - distance) / volume;
}

__device__
float smoothing_kernel_derivative(float distance)
{
  if (distance >= RADIUS_OF_INFLUENCE)
  {
    return 0.0;
  }

  float scale = 12.0 / (pow(RADIUS_OF_INFLUENCE, 4.0) * 3.141592653589);
  return (RADIUS_OF_INFLUENCE - distance) * scale;
}

__device__
float viscosity_kernel(float distance)
{
  if (distance >= RADIUS_OF_INFLUENCE)
  {
    return 0.0;
  }

  float volume = 3.141592653589 * pow(RADIUS_OF_INFLUENCE, 8.0) / 4.0;
  float value = RADIUS_OF_INFLUENCE * RADIUS_OF_INFLUENCE - distance * distance;
  return value * value * value / volume;
}

__device__
float calculate_shared_pressure(float density_a, float density_b)
{
  float pressure_a = density_to_pressure(density_a);
  float pressure_b = density_to_pressure(density_b);
  return (pressure_a + pressure_b) / 2.0;
}