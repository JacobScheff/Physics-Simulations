#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <chrono>
#include <vector>
#include <algorithm>
#include <SFML/Graphics.hpp>

// nvcc main.cu -I"C:\\Users\\jacob\\Documents\\VSC\\C++ Libraries\\SFML-2.6.1\\include" -L"C:\\Users\\jacob\\Documents\\VSC\\C++ Libraries\\SFML-2.6.1\\lib" -lsfml-graphics -lsfml-window -lsfml-system && a.exe

#define SCREEN_SIZE_X 1200
#define SCREEN_SIZE_Y 600
#define GRID_SIZE_X 80
#define GRID_SIZE_Y 40

int const TIME_BETWEEN_FRAMES = 2;                                 // The time between frames in milliseconds
float const PARTICLE_RADIUS = 1.25;                                // The radius of the particles
int const PARTICLE_AMOUNT_X = 192;                                 // The number of particles in the x direction
int const PARTICLE_AMOUNT_Y = 96;                                  // The number of particles in the y direction
int const PARTICLE_AMOUNT = PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; // The total number of particles
float const PADDING = 50.0;                                        // The padding around the screen

#define RADIUS_OF_INFLUENCE (75.0 / 4.0) // The radius of the sphere of influence. Also the radius to search for particles to calculate the density
#define TARGET_DENSITY 0.2;              // The target density of the fluid
#define PRESSURE_MULTIPLIER 500.0;       // The multiplier for the pressure force
#define GRAVITY 0.2;                     // The strength of gravity
// TODO: ADD BACK // #define LOOK_AHEAD_TIME (1.0 / 60.0); // The time to look ahead when calculating the predicted position
#define VISCOSITY 0.1;  // The viscosity of the fluid
#define DAMPENING 0.95; // How much to slow down particles when they collide with the walls
#define dt (1.0 / 8.0); // The time step

int const GRIDS_TO_CHECK[2] = {int(RADIUS_OF_INFLUENCE / SCREEN_SIZE_X * GRID_SIZE_X + 1.0), int(RADIUS_OF_INFLUENCE / SCREEN_SIZE_Y * GRID_SIZE_Y + 1.0)}; // How many grid cells to check in each direction

struct Particle
{
  sf::Vector2f position;
  sf::Vector2f velocity = {0.0, 0.0};
  float radius = PARTICLE_RADIUS;
  float density = 0.0;
  sf::Vector2f pressure_force = {0.0, 0.0};
  sf::Vector2f viscosity_force = {0.0, 0.0};
  int grid_index = -1;
};

// Grid functions
__host__ int *pos_to_grid(float x, float y)
{
  static int grid[2];
  grid[0] = (int)fmax(fmin(floor(x / SCREEN_SIZE_X * GRID_SIZE_X), GRID_SIZE_X - 1), 0);
  grid[1] = (int)fmax(fmin(floor(y / SCREEN_SIZE_Y * GRID_SIZE_Y), GRID_SIZE_Y - 1), 0);
  return grid;
}

__device__ void pos_to_grid(float x, float y, int grid[2])
{ // Pass grid as an argument
  grid[0] = (int)max(min((int)floor(x / SCREEN_SIZE_X * GRID_SIZE_X), GRID_SIZE_X - 1), 0);
  grid[1] = (int)max(min((int)floor(y / SCREEN_SIZE_Y * GRID_SIZE_Y), GRID_SIZE_Y - 1), 0);
}

__device__ __host__ int grid_to_index(int x, int y)
{
  return y * GRID_SIZE_X + x;
}

__device__ float density_to_pressure(float density)
{
  float density_error = density - TARGET_DENSITY;
  return density_error * PRESSURE_MULTIPLIER;
}

__device__ float smoothing_kernel(float distance)
{
  if (distance >= RADIUS_OF_INFLUENCE)
  {
    return 0.0;
  }

  float volume = 3.141592653589 * pow(RADIUS_OF_INFLUENCE, 4.0) / 6.0;
  return (RADIUS_OF_INFLUENCE - distance) * (RADIUS_OF_INFLUENCE - distance) / volume;
}

__device__ float smoothing_kernel_derivative(float distance)
{
  if (distance >= RADIUS_OF_INFLUENCE)
  {
    return 0.0;
  }

  float scale = 12.0 / (pow(RADIUS_OF_INFLUENCE, 4.0) * 3.141592653589);
  return (RADIUS_OF_INFLUENCE - distance) * scale;
}

__device__ float viscosity_kernel(float distance)
{
  if (distance >= RADIUS_OF_INFLUENCE)
  {
    return 0.0;
  }

  float volume = 3.141592653589 * pow(RADIUS_OF_INFLUENCE, 8.0) / 4.0;
  float value = RADIUS_OF_INFLUENCE * RADIUS_OF_INFLUENCE - distance * distance;
  return value * value * value / volume;
}

__device__ float calculate_shared_pressure(float density_a, float density_b)
{
  float pressure_a = density_to_pressure(density_a);
  float pressure_b = density_to_pressure(density_b);
  return (pressure_a + pressure_b) / 2.0;
}

// Kernel function to calculate densities
__global__ void calculate_densities(Particle *particles, int *particle_lookup, int *particle_counts, int GRIDS_TO_CHECK_X, int GRIDS_TO_CHECK_Y, int particle_amount)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= PARTICLE_AMOUNT)
    return;

  int grid[2];
  pos_to_grid(particles[index].position.x, particles[index].position.y, grid);
  float density = 0.0;

  for (int g = 0; g < (GRIDS_TO_CHECK_X * 2 + 1) * (GRIDS_TO_CHECK_Y * 2 + 1); g++)
  {
    int gx = g / (GRIDS_TO_CHECK_Y * 2 + 1) - GRIDS_TO_CHECK_X;
    int gy = g % (GRIDS_TO_CHECK_Y * 2 + 1) - GRIDS_TO_CHECK_Y;

    if (grid[0] + gx < 0 || grid[0] + gx >= GRID_SIZE_X || grid[1] + gy < 0 || grid[1] + gy >= GRID_SIZE_Y)
    {
      continue;
    }

    int first_grid_index = grid_to_index(grid[0] + gx, grid[1] + gy);
    if (first_grid_index < 0 || first_grid_index >= GRID_SIZE_X * GRID_SIZE_Y)
    {
      continue;
    }

    int starting_index = particle_lookup[first_grid_index];
    if (starting_index == -1)
    {
      continue;
    }

    int ending_index = starting_index + particle_counts[first_grid_index] - 1;
    if (ending_index >= PARTICLE_AMOUNT)
    {
      ending_index = PARTICLE_AMOUNT - 1;
    }

    float x = particles[index].position.x;
    float y = particles[index].position.y;
    for (int i = starting_index; i <= ending_index; i++)
    {
      float distance = sqrtf((particles[i].position.x - x, 2.0) * (particles[i].position.x - x, 2.0) + (particles[i].position.y - y, 2.0) * (particles[i].position.y - y, 2.0));
      if (distance < RADIUS_OF_INFLUENCE)
      {
        float influence = smoothing_kernel(distance);
        density += influence * 3.1415926f * particles[i].radius * particles[i].radius;
      }
    }
  }

  particles[index].density = density;
}

// Kernel function to calculate forces
__global__ void calculate_forces(Particle *particles, int *particle_lookup, int *particle_counts, int GRIDS_TO_CHECK_X, int GRIDS_TO_CHECK_Y, int particle_amount)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= PARTICLE_AMOUNT)
    return;

    int grid[2];
    pos_to_grid(particles[index].position.x, particles[index].position.y, grid);

    float pressure_force[2] = {0.0, 0.0};
    float viscosity_force[2] = {0.0, 0.0};

  for (int g = 0; g < (GRIDS_TO_CHECK_X * 2 + 1) * (GRIDS_TO_CHECK_Y * 2 + 1); g++)
  {
    int gx = g / (GRIDS_TO_CHECK_Y * 2 + 1) - GRIDS_TO_CHECK_X;
    int gy = g % (GRIDS_TO_CHECK_Y * 2 + 1) - GRIDS_TO_CHECK_Y;

    if (grid[0] + gx < 0 || grid[0] + gx >= GRID_SIZE_X || grid[1] + gy < 0 || grid[1] + gy >= GRID_SIZE_Y)
    {
      continue;
    }

    int first_grid_index = grid_to_index(grid[0] + gx, grid[1] + gy);
    if (first_grid_index < 0 || first_grid_index >= GRID_SIZE_X * GRID_SIZE_Y)
    {
      continue;
    }

    int starting_index = particle_lookup[first_grid_index];
    if (starting_index == -1)
    {
      continue;
    }

    int ending_index = starting_index + particle_counts[first_grid_index];

    for (int i = starting_index; i <= ending_index; i++)
    {
          // float offset[2] = {particles[i].position.x - particles[index].position.x, particles[i].position.y - particles[index].position.y};
      // float distance = sqrt(offset.x * offset.x + offset.y * offset.y);
  //     // if (distance == 0 || distance >= RADIUS_OF_INFLUENCE)
  //     // {
  //     //   continue;
  //     // }
  //     // sf::Vector2f dir = offset / distance;

  //     // float slope = smoothing_kernel_derivative(distance);
  //     // float shared_pressure = calculate_shared_pressure(particles[index].density, particles[i].density);

  //     // float pressure_multiplier = shared_pressure * slope * 3.141592653589 * particles[i].radius * particles[i].radius / max(particles[index].density, 0.000001);
  //     // sf::Vector2f local_pressure_force = dir * pressure_multiplier;

  //     // sf::Vector2f local_viscosity_force = (particles[i].velocity - particles[index].velocity) * viscosity_kernel(distance);
  //     // local_viscosity_force.x *= VISCOSITY;
  //     // local_viscosity_force.y *= VISCOSITY;

  //     // pressure_force += local_pressure_force;
  //     // viscosity_force += local_viscosity_force;
    }
  }

  particles[index].pressure_force.x = pressure_force[0];
  particles[index].pressure_force.y = pressure_force[1];
  particles[index].viscosity_force.x = viscosity_force[0];
  particles[index].viscosity_force.y = viscosity_force[1];
}

void sort(std::vector<Particle> &particles, std::vector<int> &particle_lookup, std::vector<int> &particle_counts)
{
  // Update the grid indices of the particles
  for (int i = 0; i < PARTICLE_AMOUNT; i++)
  {
    int *grid = pos_to_grid(particles[i].position.x, particles[i].position.y);
    particles[i].grid_index = grid_to_index(grid[0], grid[1]);
  }

  // Sort the particles based on grid index
  std::sort(particles.begin(), particles.end(), [](const Particle &a, const Particle &b)
            { return a.grid_index < b.grid_index; });

  // Update the particle lookup and counts
  for (int i = 0; i < GRID_SIZE_X * GRID_SIZE_Y; i++)
  {
    particle_lookup[i] = -1;
    particle_counts[i] = 0;
  }

  for (int i = 0; i < PARTICLE_AMOUNT; i++)
  {
    if (particle_lookup[particles[i].grid_index] == -1)
    {
      particle_lookup[particles[i].grid_index] = i;
    }
    particle_counts[particles[i].grid_index]++;
  }

  int currentGridIndex = -1;
  for (int i = 0; i < PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; ++i)
  {
    if (particles[i].grid_index != currentGridIndex)
    {
      particle_lookup[particles[i].grid_index] = i;
      currentGridIndex = particles[i].grid_index;
    }
    particle_counts[particles[i].grid_index]++;
  }
}

int main(void)
{
  sf::RenderWindow window(sf::VideoMode(SCREEN_SIZE_X, SCREEN_SIZE_Y), "Fluid Simulation");

  // Initialize data
  std::vector<Particle> particles(PARTICLE_AMOUNT);
  std::vector<int> particle_lookup(GRID_SIZE_X * GRID_SIZE_Y, -1);
  std::vector<int> particle_counts(GRID_SIZE_X * GRID_SIZE_Y, 0);

  for (int i = 0; i < PARTICLE_AMOUNT_X; i++)
  {
    for (int j = 0; j < PARTICLE_AMOUNT_Y; j++)
    {
      int index = i + j * PARTICLE_AMOUNT_X;
      particles[index].position = {(i + 0.5f) * (SCREEN_SIZE_X - 2.0f * PADDING) / PARTICLE_AMOUNT_X + PADDING, (j + 0.5f) * (SCREEN_SIZE_Y - 2.0f * PADDING) / PARTICLE_AMOUNT_Y + PADDING};
    }
  }

  for (int i = 0; i < GRID_SIZE_X * GRID_SIZE_Y; i++)
  {
    particle_lookup[i] = -1;
    particle_counts[i] = 0;
  }

  // Allocate Unified Memory – accessible from CPU or GPU
  Particle *d_particles;
  int *d_particle_lookup;
  int *d_particle_counts;
  hipMalloc(&d_particles, PARTICLE_AMOUNT * sizeof(Particle));
  hipMalloc(&d_particle_lookup, GRID_SIZE_X * GRID_SIZE_Y * sizeof(int));
  hipMalloc(&d_particle_counts, GRID_SIZE_X * GRID_SIZE_Y * sizeof(int));

  // Sort the particles
  sort(particles, particle_lookup, particle_counts);

  while (window.isOpen())
  {
    sf::Event event;
    while (window.pollEvent(event))
    {
      if (event.type == sf::Event::Closed)
        window.close();
    }

    // Get start time
    auto start = std::chrono::high_resolution_clock::now();

    // Copy data to the GPU
    hipMemcpy(d_particles, particles.data(), PARTICLE_AMOUNT * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_particle_lookup, particle_lookup.data(), GRID_SIZE_X * GRID_SIZE_Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_particle_counts, particle_counts.data(), GRID_SIZE_X * GRID_SIZE_Y * sizeof(int), hipMemcpyHostToDevice);

    // Get the number of blocks and threads
    int blockSize = 256;
    int numBlocks = (PARTICLE_AMOUNT + blockSize - 1) / blockSize;

    // Calculate densities
    calculate_densities<<<numBlocks, blockSize>>>(d_particles, d_particle_lookup, d_particle_counts, GRIDS_TO_CHECK[0], GRIDS_TO_CHECK[1], PARTICLE_AMOUNT);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
      std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    // // Calculate forces
    // calculate_forces<<<numBlocks, blockSize>>>(d_particles, d_particle_lookup, d_particle_counts, GRIDS_TO_CHECK[0], GRIDS_TO_CHECK[1], PARTICLE_AMOUNT);
    
    // // Wait for GPU to finish before accessing on host
    // hipDeviceSynchronize();
    // err = hipGetLastError();
    // if (err != hipSuccess)
    // {
    //   std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    // }

    // Copy data back from GPU
    hipMemcpy(particles.data(), d_particles, PARTICLE_AMOUNT * sizeof(Particle), hipMemcpyDeviceToHost);

    // Print end time in ms
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    std::cout << "Elapsed time in milliseconds : " << elapsed.count() << " ms" << std::endl;

    // NOTE: DRAWING IS VERY SLOW
    std::vector<sf::CircleShape> circles;
    for (int i = 0; i < PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; ++i)
    {
      sf::CircleShape circle(PARTICLE_RADIUS);
      circle.setFillColor(sf::Color::Blue);
      circle.setPosition(particles[i].position);
      circles.push_back(circle);
    }

    window.clear();
    for (const auto &circle : circles)
    {
      window.draw(circle);
    }
    window.display();

    // Wait for TIME_BETWEEN_FRAMES
    sf::sleep(sf::milliseconds(TIME_BETWEEN_FRAMES));
  }

  // Free memory
  hipFree(d_particles);
  hipFree(d_particle_lookup);
  hipFree(d_particle_counts);

  std::cout << "Hello, World!" << std::endl;

  return 0;
}