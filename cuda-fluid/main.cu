
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <array>

// Kernel function to calculate densities
__global__
void calculate_densities(float **positions, float *densities, float *radii, int PARTICLE_AMOUNT) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= PARTICLE_AMOUNT) return;

  float density = 0.0;
  
  densities[index] = density;
}

int main(void)
{
  std::array<int, 2> const SCREEN_SIZE = {800, 400}; // The size of the screen
  int const TIME_BETWEEN_FRAMES = 2;
  std::array<int, 2> const GRID_SIZE = {80, 40}; // How many grid cells to divide the screen into

  float const PARTICLE_RADIUS = 1.25; // The radius of the particles
  int const PARTICLE_AMOUNT_X = 192; // The number of particles in the x direction
  int const PARTICLE_AMOUNT_Y = 96; // The number of particles in the y direction
  int const PARTICLE_AMOUNT = PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; // The total number of particles
  float const PADDING = 50.0; // The padding around the screen

  // Initialize data
  float **positions, **velocities, **pressure_force, **viscosity_force;
  float *densities, *radii;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&positions, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&velocities, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&pressure_force, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&viscosity_force, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&densities, PARTICLE_AMOUNT * sizeof(float));
  hipMallocManaged(&radii, PARTICLE_AMOUNT * sizeof(float));

  // Initialize data
  for (int i = 0; i < PARTICLE_AMOUNT; i++) {
    positions[i] = new float[2];
    velocities[i] = new float[2];
    pressure_force[i] = new float[2];
    viscosity_force[i] = new float[2];
    positions[i][0] = (i + 0.5) * (SCREEN_SIZE[0] - 2.0 * PADDING) / PARTICLE_AMOUNT_X + PADDING;
    positions[i][1] = (i + 0.5) * (SCREEN_SIZE[1] - 2.0 *   PADDING) / PARTICLE_AMOUNT_Y + PADDING;
    velocities[i][0] = 0.0;
    velocities[i][1] = 0.0;
    densities[i] = 0.0;
    radii[i] = PARTICLE_RADIUS;
    pressure_force[i][0] = 0.0;
    pressure_force[i][1] = 0.0;
    viscosity_force[i][0] = 0.0;
    viscosity_force[i][1] = 0.0;
  }

  // Get start time
  auto start = std::chrono::high_resolution_clock::now();

  // Get the number of blocks and threads
  int blockSize = 256;
  int numBlocks = (PARTICLE_AMOUNT + blockSize - 1) / blockSize;

  // Calculate densities
  calculate_densities<<<numBlocks, blockSize>>>(positions, densities, radii, PARTICLE_AMOUNT);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Print end time in ms
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> elapsed = end - start;
  std::cout << "Elapsed time in milliseconds : " << elapsed.count() << " ms" << std::endl;

  // Free memory
  hipFree(positions);
  hipFree(velocities);
  hipFree(pressure_force);
  hipFree(viscosity_force);
  hipFree(densities);
  hipFree(radii);

  std::cout << "Hello, World!" << std::endl;
  
  return 0;
}
