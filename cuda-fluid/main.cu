
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <array>
#include <vector>

std::array<int, 2> const SCREEN_SIZE = {800, 400}; // The size of the screen
std::array<int, 2> const GRID_SIZE = {80, 40};     // How many grid cells to divide the screen into

int const TIME_BETWEEN_FRAMES = 2;
float const PARTICLE_RADIUS = 1.25;                                // The radius of the particles
int const PARTICLE_AMOUNT_X = 192;                                 // The number of particles in the x direction
int const PARTICLE_AMOUNT_Y = 96;                                  // The number of particles in the y direction
int const PARTICLE_AMOUNT = PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; // The total number of particles
float const PADDING = 50.0;                                        // The padding around the screen

// Grid functions
__host__
std::array<int, 2> pos_to_grid(float x, float y)
{
  return {
      (int)fmax(fmin(floor(x / SCREEN_SIZE[0] * GRID_SIZE[0]), GRID_SIZE[0] - 1), 0),
      (int)fmax(fmin(floor(y / SCREEN_SIZE[1] * GRID_SIZE[1]), GRID_SIZE[1] - 1), 0)};
}

__host__
int grid_to_index(int x, int y)
{
  return y * GRID_SIZE[0] + x;
}

// Kernel function to calculate densities
__global__ void calculate_densities(float **positions, float *densities, float *radii, int PARTICLE_AMOUNT)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= PARTICLE_AMOUNT)
    return;

  float density = 0.0;

  // TODO: Finish code

  densities[index] = density;
}

void sort(float **positions, float **velocities, float *radii, float *densities, float **pressure_force, float **viscosity_force, int *particle_lookup, int *particle_counts)
{
  // Map all particles to their grid cell
  std::vector<std::vector<std::vector<int>>> index_map(GRID_SIZE[0], std::vector<std::vector<int>>(GRID_SIZE[1], std::vector<int>()));
  for (int i = 0; i < PARTICLE_AMOUNT; i++)
  {
    std::array<int, 2> grid = pos_to_grid(positions[i][0], positions[i][1]);
    index_map[grid[0]][grid[1]].push_back(i);
  }

  // Create a new list of particles
  float **new_positions = new float *[PARTICLE_AMOUNT];
  float **new_velocities = new float *[PARTICLE_AMOUNT];
  float *new_radii = new float[PARTICLE_AMOUNT];
  float *new_densities = new float[PARTICLE_AMOUNT];
  float **new_pressure_force = new float *[PARTICLE_AMOUNT];
  float **new_viscosity_force = new float *[PARTICLE_AMOUNT];

  // Iterate over all grid cells
  for (int i = 0; i < GRID_SIZE[0]; i++)
  {
    for (int j = 0; j < GRID_SIZE[1]; j++)
    {
      int grid_index = i + j * GRID_SIZE[0];
      int index = -1;

      // Iterate over all particles in the grid cell
      for (int k = 0; k < index_map[i][j].size(); k++)
      {
        int particle_index = index_map[i][j][k];
        new_positions[particle_index] = positions[particle_index];
        new_velocities[particle_index] = velocities[particle_index];
        new_radii[particle_index] = radii[particle_index];
        new_densities[particle_index] = densities[particle_index];
        new_pressure_force[particle_index] = pressure_force[particle_index];
        new_viscosity_force[particle_index] = viscosity_force[particle_index];

        if (index == -1)
        {
          index = particle_index;
        }
        particle_counts[grid_index]++;
      }

      particle_lookup[grid_index] = index;
    }
  }

  positions = new_positions;
  velocities = new_velocities;
  radii = new_radii;
  densities = new_densities;
  pressure_force = new_pressure_force;
  viscosity_force = new_viscosity_force;
}

int main(void)
{
  // Initialize data
  float **positions, **velocities, **pressure_force, **viscosity_force;
  float *densities, *radii;
  int *particle_lookup, *particle_counts;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&positions, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&velocities, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&pressure_force, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&viscosity_force, PARTICLE_AMOUNT * sizeof(float *));
  hipMallocManaged(&densities, PARTICLE_AMOUNT * sizeof(float));
  hipMallocManaged(&radii, PARTICLE_AMOUNT * sizeof(float));
  hipMallocManaged(&particle_lookup, GRID_SIZE[0] * GRID_SIZE[1] * sizeof(int));
  hipMallocManaged(&particle_counts, GRID_SIZE[0] * GRID_SIZE[1] * sizeof(int));

  // Initialize data
  for (int i = 0; i < PARTICLE_AMOUNT; i++)
  {
    positions[i] = new float[2];
    velocities[i] = new float[2];
    pressure_force[i] = new float[2];
    viscosity_force[i] = new float[2];
    positions[i][0] = (i + 0.5) * (SCREEN_SIZE[0] - 2.0 * PADDING) / PARTICLE_AMOUNT_X + PADDING;
    positions[i][1] = (i + 0.5) * (SCREEN_SIZE[1] - 2.0 * PADDING) / PARTICLE_AMOUNT_Y + PADDING;
    velocities[i][0] = 0.0;
    velocities[i][1] = 0.0;
    densities[i] = 0.0;
    radii[i] = PARTICLE_RADIUS;
    pressure_force[i][0] = 0.0;
    pressure_force[i][1] = 0.0;
    viscosity_force[i][0] = 0.0;
    viscosity_force[i][1] = 0.0;

    if (i < GRID_SIZE[0] * GRID_SIZE[1])
    {
      particle_lookup[i] = -1;
      particle_counts[i] = 0;
    }
  }
  
  // Sort the particles
  sort(positions, velocities, radii, densities, pressure_force, viscosity_force, particle_lookup, particle_counts);

  // Get start time
  auto start = std::chrono::high_resolution_clock::now();

  // Get the number of blocks and threads
  int blockSize = 256;
  int numBlocks = (PARTICLE_AMOUNT + blockSize - 1) / blockSize;

  // Calculate densities
  calculate_densities<<<numBlocks, blockSize>>>(positions, densities, radii, PARTICLE_AMOUNT);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Print end time in ms
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> elapsed = end - start;
  std::cout << "Elapsed time in milliseconds : " << elapsed.count() << " ms" << std::endl;

  // Free memory
  hipFree(positions);
  hipFree(velocities);
  hipFree(pressure_force);
  hipFree(viscosity_force);
  hipFree(densities);
  hipFree(radii);

  std::cout << "Hello, World!" << std::endl;

  return 0;
}