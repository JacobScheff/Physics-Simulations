
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <array>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  std::array<int, 2> const SCREEN_SIZE = {800, 400}; // The size of the screen
  int const TIME_BETWEEN_FRAMES = 2;
  std::array<int, 2> const GRID_SIZE = {80, 40}; // How many grid cells to divide the screen into

  float const PARTICLE_RADIUS = 1.25; // The radius of the particles
  int const PARTICLE_AMOUNT_X = 192; // The number of particles in the x direction
  int const PARTICLE_AMOUNT_Y = 96; // The number of particles in the y direction
  int const PARTICLE_AMOUNT = PARTICLE_AMOUNT_X * PARTICLE_AMOUNT_Y; // The total number of particles
  float const PADDING = 50.0; // The padding around the screen

  // Initialize data
  float **positions, **velocities, **pressure_force, **viscosity_force;
  float *densities, *radii;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&positions, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&velocities, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&pressure_force, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&viscosity_force, PARTICLE_AMOUNT * sizeof(float*));
  hipMallocManaged(&densities, PARTICLE_AMOUNT * sizeof(float));
  hipMallocManaged(&radii, PARTICLE_AMOUNT * sizeof(float));

  // Initialize data
  for (int i = 0; i < PARTICLE_AMOUNT; i++) {
    positions[i] = new float[2];
    velocities[i] = new float[2];
    pressure_force[i] = new float[2];
    viscosity_force[i] = new float[2];
    positions[i][0] = (i + 0.5) * (SCREEN_SIZE[0] - 2.0 * PADDING) / PARTICLE_AMOUNT_X + PADDING;
    positions[i][1] = (i + 0.5) * (SCREEN_SIZE[1] - 2.0 * PADDING) / PARTICLE_AMOUNT_Y + PADDING;
    velocities[i][0] = 0.0;
    velocities[i][1] = 0.0;
    densities[i] = 0.0;
    radii[i] = PARTICLE_RADIUS;
    pressure_force[i][0] = 0.0;
    pressure_force[i][1] = 0.0;
    viscosity_force[i][0] = 0.0;
    viscosity_force[i][1] = 0.0;
  }

  // Get start time
  auto start = std::chrono::high_resolution_clock::now();

  // // // Run kernel on the GPU
  // // int blockSize = 256;
  // // int numBlocks = (N + blockSize - 1) / blockSize;
  // // add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Get end time
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<float> duration = end - start;
  std::cout << "Time: " << duration.count() << "s" << std::endl;

  // Free memory
  hipFree(positions);
  hipFree(velocities);
  hipFree(pressure_force);
  hipFree(viscosity_force);
  hipFree(densities);
  hipFree(radii);

  std::cout << "Hello, World!" << std::endl;
  
  return 0;
}
